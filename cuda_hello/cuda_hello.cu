
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void cuda_say_hello() {
	printf("Hello, Cuda! %d\n", threadIdx.x);
}


int main() {
	printf("Hello, Cpu!\n");

	cuda_say_hello<<<1, 1>>>();

	hipError_t cudaerr = hipDeviceSynchronize();
	if (cudaerr != hipSuccess) {
		printf("kernel launched with error %s\n", hipGetErrorString(cudaerr));
	}
	return 0;
}
