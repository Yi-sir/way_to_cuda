
#include <hip/hip_runtime.h>
#include <iostream>

/*
 * @param n: batch size
 * @param c: 通道数
 * @param h: 输入数据高
 * @param w: 输入数据宽
 * @param k: 卷积核数量
 * @param r: 卷积核高
 * @param s: 卷积核宽
 * @param out_h: 输出数据高
 * @param out_w: 输出数据宽
 * @param u: 卷积在高方向上的步长
 * @param v: 卷积在宽方向上的步长
 * @param p: 卷积在高方向上的补边
 * @param q: 卷积在宽方向上的补边
 * @param in: 输入数据
 * @param weight: 卷积核
 * @param out: 输出数据
 */
__global__ void naive_conv2d_kernel(int n, int c, int h, int w, int k, int r,
                                    int s, int out_h, int out_w, int u, int v,
                                    int p, int q, float* in, float* weight,
                                    float* out) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int z = blockIdx.z;

  if (x >= out_h * out_w || y >= k || z >= n) {
    return;
  }

  int pos_out_h = x / out_w;
  int pos_out_w = x % out_w;

  float sum = 0.0;

  int pos_ori_h = pos_out_h * u - p;
  int pos_ori_w = pos_out_w * v - q;

  int in_offset = z * c * h * w + pos_ori_h * w + pos_ori_w;
  int weight_offset = y * c * r * s;
  int in_channel_offset = h * w;
  int weight_channel_offset = r * s;

  for (int i = 0; i < r; ++i) {
    for (int j = 0; j < s; ++j) {
      int pos_real_h = pos_ori_h + i;
      int pos_real_w = pos_ori_w + j;
      if (pos_real_h < 0 || pos_real_w < 0 || pos_real_h >= h ||
          pos_real_w >= w) {
        continue;
      }
      for (int channel = 0; channel < c; ++channel) {
        sum +=
            in[in_offset + channel * in_channel_offset + i * w + j] *
            weight[weight_offset + channel * weight_channel_offset + i * s + j];
      }
    }
  }
  int out_offset = z * k * out_w * out_h + y * out_h * out_w + x;
  out[out_offset] = sum;
}

void naive_conv2d_cpu(int n, int c, int h, int w, int k, int r, int s,
                      int out_h, int out_w, int u, int v, int p, int q,
                      float* in, float* weight, float* out) {
  // Out(N_i, C_out_j) = sigma_k=0_k=C_in-1(weight(C_out_j, k) * input(N_i, k))
  // batch loop
  for (int n_i = 0; n_i < n; ++n_i) {
    int n_input_offset = n_i * c * h * w;
    // output channel loop

    for (int k_i = 0; k_i < k; ++k_i) {
      int k_offset = k_i * c * r * s;

      // output pixel loop
      for (int out_i = 0; out_i < out_h; ++out_i) {
        for (int out_j = 0; out_j < out_w; ++out_j) {
          // 输出[out_i, out_j]对应输入的某个区域的起始点
          int input_start_i = out_i * u - p;
          int input_start_j = out_j * v - q;
          double sum = 0.0;

          for (int c_i = 0; c_i < c; ++c_i) {
            int c_kernel_offset = c_i * r * s;
            int c_input_offset = c_i * h * w;

            for (int weight_i = 0; weight_i < r; ++weight_i) {
              for (int weight_j = 0; weight_j < s; ++weight_j) {
                // 权重的位置
                int weight_offset =
                    k_offset + c_kernel_offset + weight_i * s + weight_j;
                // 原图区域里当前权重对应点的坐标
                int input_current_i = input_start_i + weight_i;
                int input_current_j = input_start_j + weight_j;

                if (input_current_i >= 0 && input_current_j >= 0 &&
                    input_current_i < h && input_current_j < w) {
                  // 原图数据的位置
                  int input_offset = n_input_offset + c_input_offset +
                                     input_current_i * w + input_current_j;
                  sum += (double)in[input_offset] * weight[weight_offset];
                }
              }
            }
          }
          int output_offset = n_i * k * out_h * out_w + k_i * out_h * out_w +
                              out_i * out_w + out_j;
          out[output_offset] = (float)sum;
        }
      }
    }
  }
}

void conv2d_cpu(float* in, float* pwei, float* out, int n, int c, int h, int w,
                int k, int r, int s, int u, int v, int p, int q) {
  int out_h = (h + 2 * p - r) / u + 1;
  int out_w = (w + 2 * q - s) / v + 1;

  for (int n_num = 0; n_num < n; n_num++) {
    for (int k_num = 0; k_num < k; k_num++) {
      for (int i = 0; i < out_h; i++) {
        for (int j = 0; j < out_w; j++) {
          double sum = 0.0;
          int pos_h = i * u - p;
          int pos_w = j * v - q;

          for (int c_num = 0; c_num < c; c_num++) {
            for (int kh_num = 0; kh_num < r; kh_num++) {
              for (int kwNum = 0; kwNum < s; kwNum++) {
                int pos_ori_h = pos_h + kh_num;
                int pos_ori_w = pos_w + kwNum;
                if (pos_ori_w >= 0 && pos_ori_h >= 0 && pos_ori_w < w &&
                    pos_ori_h < h) {
                  sum += (double)(in[n_num * c * h * w + c_num * (w * h) +
                                     pos_ori_h * w + pos_ori_w] *
                                  pwei[k_num * r * s * c + c_num * r * s +
                                       kh_num * s + kwNum]);
                }
              }
            }
          }

          out[n_num * k * out_h * out_w + k_num * out_h * out_w + i * out_w +
              j] = (float)sum;
        }
      }
    }
  }
}

int main() {
  // 定义输入数据和卷积核的尺寸
  const int n = 2;                            // batch size
  const int c = 2;                            // 通道数
  const int h = 10;                           // 数据高
  const int w = 10;                           // 数据宽
  const int k = 5;                            // 卷积核数量
  const int r = 3;                            // 卷积核高
  const int s = 3;                            // 卷积核宽
  const int u = 1;                            // 卷积在高方向上的步长
  const int v = 1;                            // 卷积在宽方向上的步长
  const int p = 0;                            // 卷积在高方向上的补边
  const int q = 0;                            // 卷积在宽方向上的补边
  const int out_h = (h - r + 2 * p) / u + 1;  // 输出高
  const int out_w = (w - s + 2 * q) / v + 1;  // 输出宽

  float *in_device, *weight_device, *out_device;

  float* in = new float[n * c * h * w];
  float* weight = new float[k * c * r * s];
  float* out = new float[n * k * out_h * out_w];
  float* out_cpu = new float[n * k * out_h * out_w];

  hipMalloc(&in_device, sizeof(float) * n * c * h * w);
  hipMalloc(&weight_device, sizeof(float) * k * c * r * s);
  hipMalloc(&out_device, sizeof(float) * n * k * out_h * out_w);

  for (int i = 0; i < n * c * h * w; ++i) {
    in[i] = (float)rand() / RAND_MAX;
  }
  for (int i = 0; i < k * c * r * s; ++i) {
    weight[i] = (float)rand() / RAND_MAX;
  }

  hipMemcpy(in_device, in, sizeof(float) * n * c * h * w,
             hipMemcpyHostToDevice);
  hipMemcpy(weight_device, weight, sizeof(float) * k * c * r * s,
             hipMemcpyHostToDevice);
  hipMemcpy(out_device, out, sizeof(float) * n * k * out_h * out_w,
             hipMemcpyHostToDevice);

  const int blockDim_x = 16;
  const int blockDim_y = 16;

  const int gridDim_x = (out_h * out_w + blockDim_x - 1) / blockDim_x;
  const int gridDim_y = (k + blockDim_y - 1) / blockDim_y;

  dim3 blockDim{blockDim_x, blockDim_y};
  dim3 gridDim{gridDim_x, gridDim_y, n};

  naive_conv2d_kernel<<<gridDim, blockDim>>>(n, c, h, w, k, r, s, out_h, out_w,
                                             u, v, p, q, in_device,
                                             weight_device, out_device);
  hipDeviceSynchronize();

  hipMemcpy(out, out_device, sizeof(float) * n * k * out_h * out_w,
             hipMemcpyDeviceToHost);

  naive_conv2d_cpu(n, c, h, w, k, r, s, out_h, out_w, u, v, p, q, in, weight,
                   out_cpu);
  bool pass = true;
  for (int i = 0; i < n * k * out_h * out_w; ++i) {
    if (abs(out[i] - out_cpu[i]) > 1e-5) {
      pass = false;
      std::cout << "Verification failed at " << i << "!" << std::endl;
      std::cout << "GPU: " << out_cpu[i] << " CPU: " << out[i] << std::endl;
      break;
    }
  }
  if (pass) {
    std::cout << "Verification passed!" << std::endl;
  }

  delete in;
  delete weight;
  delete out;
  delete out_cpu;

  hipFree(in_device);
  hipFree(weight_device);
  hipFree(out_device);

  return 0;
}