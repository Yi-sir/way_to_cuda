#include <hip/hip_runtime.h>

#include <iostream>

#define CEIL_DIV(M, N) (((M) + (N) - 1) / (N))

void sgemm_naive_cpu(float* A, float* B, float* C, int M, int N, int K) {
  for (int x = 0; x < M; ++x) {
    for (int y = 0; y < N; ++y) {
      float sum = 0.0f;
      for (int i = 0; i < K; ++i) {
        sum += A[x * K + i] * B[i * N + y];
      }
      C[x * N + y] = sum;
    }
  }
}

template <const int BLOCK_SIZE>
__global__ void sgemm_shared_mem_kernel(float* A, float* B, float* C, int M,
                                        int N, int K) {
  // 当前block计算的是C上坐标为[c_row, c_col]的block
  const uint c_row = blockIdx.x;
  const uint c_col = blockIdx.y;

  // 线程块内共享
  __shared__ float A_shared[BLOCK_SIZE * BLOCK_SIZE];
  __shared__ float B_shared[BLOCK_SIZE * BLOCK_SIZE];

  // 当前thread计算的是C上当前block内坐标为[thread_row, thread_col]的值
  const uint thread_row = threadIdx.x / BLOCK_SIZE;
  const uint thread_col = threadIdx.x % BLOCK_SIZE;

  // A: 指针移动到目标block对应行的开头
  A += c_row * BLOCK_SIZE * K;
  // B: 指针移动到目标block对应列的开头
  B += c_col * BLOCK_SIZE;
  // C: 指针移动到目标block的开头
  C += c_row * BLOCK_SIZE * N + c_col * BLOCK_SIZE;

  float tmp = 0.0f;
  // 总共计算K次乘法，循环里每次计算BLOCK_SIZE个
  for (int i = 0; i < K; i += BLOCK_SIZE) {
    // 此时，A已经移动到了某个BLOCK的开头
    // 这里是每个线程都在搬数据
    // 所有线程的工作结果是：把A上一个BLOCK的内容搬到A_shared上
    A_shared[thread_row * BLOCK_SIZE + thread_col] =
        A[thread_row * K + thread_col];
    // B 同理
    B_shared[thread_row * BLOCK_SIZE + thread_col] =
        B[thread_row * N + thread_col];

    __syncthreads();

    // 等待每个线程都搬完数据之后，每个线程开始算自己负责的点[c_row, c_col]
    // 每次需要取BLOCK中一行/一列的内容，算BLOCK_SIZE次乘法
    for (int j = 0; j < BLOCK_SIZE; ++j) {
      tmp += A_shared[thread_row * BLOCK_SIZE + j] *
             B_shared[j * BLOCK_SIZE + thread_col];
    }

    __syncthreads();

    // 一个BLOCK计算结束，BLOCK内每个坐标的结果都计算了一部分
    // 挪到下一个BLOCK继续
    A += BLOCK_SIZE;
    B += BLOCK_SIZE * N;
  }
  // C的指针已经移动到了目标block的开头
  // 所以这里计算的坐标就是实际坐标
  C[thread_row * N + thread_col] = tmp;
}

void run_sgemm_shared_memory(float* A, float* B, float* C, int m, int n,
                             int k) {
  const int BLOCKSIZE = 32;
  dim3 block_size(BLOCKSIZE * BLOCKSIZE);
  dim3 grid_size(CEIL_DIV(m, BLOCKSIZE), CEIL_DIV(n, BLOCKSIZE));
  sgemm_shared_mem_kernel<BLOCKSIZE>
      <<<grid_size, block_size>>>(A, B, C, m, n, k);
}

void randomize_matrix(float* mat, int N) {
  for (int i = 0; i < N; i++) {
    mat[i] = rand() % 100;
  }
}

int main() {
  int m = 256, n = 256, k = 256;

  float* A = new float[m * k];
  float* B = new float[k * n];
  float* C = new float[m * n];

  float* C_ref = new float[m * n];

  randomize_matrix(A, m * k);
  randomize_matrix(B, k * n);

  float *d_A, *d_B, *d_C;

  hipMalloc((void**)&d_A, m * k * sizeof(float));
  hipMalloc((void**)&d_B, n * k * sizeof(float));
  hipMalloc((void**)&d_C, m * n * sizeof(float));

  hipMemcpy(d_A, A, m * k * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, n * k * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_C, C, m * n * sizeof(float), hipMemcpyHostToDevice);

  run_sgemm_shared_memory(d_A, d_B, d_C, m, n, k);

  hipMemcpy(C, d_C, m * n * sizeof(float), hipMemcpyDeviceToHost);

  sgemm_naive_cpu(A, B, C_ref, m, n, k);

  for (int i = 0; i < m * n; ++i) {
    if (C[i] != C_ref[i]) {
      std::cout << "Mismatch! [" << i / n << ", " << i % n << "]" << std::endl;
    }
  }

  delete A;
  delete B;
  delete C;
  delete C_ref;

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  std::cout << "Success" << std::endl;

  return 0;
}